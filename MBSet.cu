#include "hip/hip_runtime.h"
/* 
 * File:   MBSet.cu
 * 
 * Created on June 24, 2012
 * 
 * Purpose:  This program displays Mandelbrot set using the GPU via CUDA and
 * OpenGL immediate mode.
 * 
 */

#include <iostream>
#include <stack>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>
#include "Complex.cu"
#include <GL/freeglut.h>
#include <GL/glut.h>
#include <math.h>
#include <GL/glext.h>
#include <GL/gl.h>
#include <GL/glu.h>

// Size of window in pixels, both width and height
#define WINDOW_DIM            512
#define NUM_THREADS 32
using namespace std;

// Initial screen coordinates, both host and device.
// x goes from -2 to 1 and y oges from -1.2 to 1.8
Complex minC(-2.0, -1.2);
Complex maxC(1.0, 1.8);
Complex* dev_minC;
Complex* dev_maxC;
int* dev_MArr;
int MArr[512*512];
const int maxIt = 2000; // Msximum Iterations

double coordinateReal(double x){//calculate real part of coordinate from x value
return (minC.r + x*(maxC.r-minC.r)/(511));
}

double coordinateImaginary(double y){
return (maxC.i - y*(maxC.i-minC.i)/(511));
}
// Define the RGB Class
class RGB
{
public:
  RGB()
    : r(0), g(0), b(0) {}
  RGB(double r0, double g0, double b0)
    : r(r0), g(g0), b(b0) {}
public:
  double r;
  double g;
  double b;
};

RGB* colors = 0; // Array of color values

void InitializeColors()
{
  colors = new RGB[maxIt + 1];
  for (int i = 0; i < maxIt; ++i)
    {
      if (i < 5)
        { // Try this.. just white for small it counts
          colors[i] = RGB(1, 1, 1);
        }
      else
        {
          colors[i] = RGB(drand48(), drand48(), drand48());
        }
    }
  colors[maxIt] = RGB(); // black
}

__global__ void calculateMandlebrot(Complex* dev_minC, Complex* dev_maxC, int* dev_MArr){
  int n;
  int id = threadIdx.x + blockIdx.x * blockDim.x; //what pixel should I work on
  int x = id%512; //what is x position
  int y = id/512; //what is y position
  double coordImaginary = (dev_maxC->i) - y*((dev_maxC->i)-(dev_minC->i))/(511);
  double coordReal = (dev_minC->r) + x*((dev_maxC->r)-(dev_minC->r))/(511);
  Complex cCoord = Complex(coordReal, coordImaginary);
  Complex zCoord = Complex(coordReal, coordImaginary);
  for(n=0;n<maxIt;++n){
    if(zCoord.magnitude2() >  4.0)
      break;
    zCoord = (zCoord * zCoord) + cCoord;
  }  
  dev_MArr[id] = n;
/*for( int  y=0; y<WINDOW_DIM; ++y){
    double coordIm = coordinateImaginary(y);
    for( int x=0;  x<WINDOW_DIM; ++x){
      double coordReal = coordinateReal(x);
      Complex cCoord = Complex(coordReal,coordIm);
      bool isInside = true;
      Complex zCoord = Complex(coordReal,coordIm);
      for( n=0;n<=maxIt;++n){//calculate whether it is in the set
	if(zCoord.magnitude2() > 2.0)
	  {
	    isInside = false;
	    break;
	  }
	// z = z^2 + c
	zCoord = (zCoord * zCoord) + cCoord;
      }
      MArr[x][y] = n; //set array point to iteration count.
    }
  }*/
}

void displayMandlebrot(){
  //cout << "hello from displayMandlebrot" << endl;
  glBegin(GL_POINTS);
  for(int x =0; x< WINDOW_DIM; x++){
    for(int y=0; y<WINDOW_DIM; y++){
      //draw it based on iteration
      int pix = y*512 + x;
      int iterationCount = MArr[pix];
      glColor3f(colors[iterationCount].r,colors[iterationCount].g,colors[iterationCount].b);
      glVertex2f(x,y);  
    }
  }
  glEnd();
}
void display(void){
  //cout << "hello from display ()" << endl;
  glClear(GL_COLOR_BUFFER_BIT);
  glClear(GL_DEPTH_BUFFER_BIT);
  displayMandlebrot();
  glutSwapBuffers();
}

void init(){
  glShadeModel(GL_FLAT);
  glViewport(0,0,WINDOW_DIM, WINDOW_DIM);
  //drawMandlebrot();
}

void getReadyForCalcMandlebrot(){
  //allocate space for device copies
  hipMalloc((void**)&dev_MArr, WINDOW_DIM * WINDOW_DIM * sizeof(int));
  hipMalloc((void**)&dev_minC, sizeof(Complex));
  hipMalloc((void**)&dev_maxC, sizeof(Complex));
  //copy inputs to device
  hipMemcpy(dev_minC, &minC, sizeof(Complex), hipMemcpyHostToDevice);
  hipMemcpy(dev_maxC, &maxC, sizeof(Complex), hipMemcpyHostToDevice);
  //launch calculateMandlebrot() kernel
  calculateMandlebrot<<<WINDOW_DIM * WINDOW_DIM / NUM_THREADS, NUM_THREADS >>>(dev_minC, dev_maxC, dev_MArr);
  //copy result back to host
  hipMemcpy(MArr, dev_MArr, WINDOW_DIM * WINDOW_DIM * sizeof(int), hipMemcpyDeviceToHost);
  //free
  hipFree(dev_minC); hipFree(dev_maxC); hipFree(dev_MArr);
}


int main(int argc, char** argv)
{ getReadyForCalcMandlebrot();
  // Initialize OPENGL here
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
  glutInitWindowSize(WINDOW_DIM,WINDOW_DIM);
  glutInitWindowPosition(100,100);
  glutCreateWindow("Mandlebrot");
  init();
  glViewport(0,0, (GLsizei) 512, (GLsizei) 512);
  glMatrixMode (GL_PROJECTION);
  glLoadIdentity();
  glOrtho(0, WINDOW_DIM, 0, WINDOW_DIM, -WINDOW_DIM, WINDOW_DIM);
  // Set up necessary host and device buffers
  // set up the opengl callbacks for display, mouse and keyboard
  glutDisplayFunc(display);
  glutIdleFunc(display);
  // Calculate the interation counts
  // Grad students, pick the colors for the 0 .. 1999 iteration count pixels
  InitializeColors();
  glutMainLoop(); // THis will callback the display, keyboard and mouse
  return 0;
  
}
